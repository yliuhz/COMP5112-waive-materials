
#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <omp.h>
#include <cstring>
#include <iostream>
#include <vector>

using namespace std;

struct B
{
    int bb = -1;
};

struct A
{
    B *b;
    int aa = 1;
};

inline void setm(A *ha, A **da)
{
    hipMalloc((void**)da, sizeof(A));
    hipMalloc((void**)&((*da)->b), sizeof(B) * 50000);
    hipMemcpy((*da)->b, ha->b, sizeof(B) * 50000, hipMemcpyHostToDevice);
}

inline void freem(A *ha, A **da)
{
    hipFree((*da)->b);
    hipFree((*da));
}

int main(void)
{
    A *ha, *da;
    
    ha = new A();
    ha->b = new B[5];

    setm(ha, &da);
    freem(ha, &da);
    delete[] ha->b;
    delete ha;
}
